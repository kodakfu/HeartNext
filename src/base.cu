#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include"base.h"
#include<stdio.h>
#include<assert.h>
void fill_n_v(device_vector_double& data , int n, typename device_vector_double::value_type v){
	 data.resize(n,v);
}
void fill_n_v(device_vector_int& data , int n, typename device_vector_int::value_type v){
	 data.resize(n,v);
}
__global__ void NotGetDoubleDataByIndexKernel(double* src, int* index, double* dst, int maxId)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= maxId) return;
	int theIndex = index[id];
	dst[theIndex] = src[id];
}
__global__ void GetDoubleDataByIndexKernel(double* src, int* index, double* dst, int maxId)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= maxId) return;
	int theIndex = index[id];
	src[id] = dst[theIndex];
}

void do_MapDoubleDataByIndex(double* src, int* index, double* dst, int number, bool get)
{
	int blockDim = 128;
	int gridDim = number/blockDim;
	gridDim = gridDim * blockDim < number ? gridDim + 1 : gridDim;
	if(get)
		GetDoubleDataByIndexKernel<<< gridDim, blockDim >>>(src, index, dst, number);
	else
		NotGetDoubleDataByIndexKernel<<< gridDim, blockDim >>>(src, index, dst, number);
	hipDeviceSynchronize();
}

__global__ void VoltKernel(double* new_volt, double* volt, double* current, int* xpos, int* xneg,
						   int* ypos, int* yneg, int* zpos, int* zneg,
						   double DD, double dt, int maxId)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= maxId) return;
	double base = volt[id];
	double baseX2 = 2 * base;
	double vxpos = volt[ xpos[id] ];
	double vxneg = volt[ xneg[id] ];
	double xdiff = vxpos + vxneg - baseX2;
	double vypos = volt[ ypos[id] ];
	double vyneg = volt[ yneg[id] ];
	double ydiff = vypos + vyneg - baseX2;
	double vzpos = volt[ zpos[id] ];
	double vzneg = volt[ zneg[id] ];
	double zdiff = vzpos + vzneg - baseX2;
	double dv = ( DD * (xdiff + ydiff + zdiff) - current[id] ) * dt;
	/*
	if(id == 1282860) {
		printf("%lf %lf %lf %lf %lf %lf\n", vxpos, vxneg, vypos, vyneg, vzpos, vzneg);
		printf("%lf, %lf, %lf\n", xdiff, ydiff, zdiff);
		printf("%lf + %lf =  %lf at VoltKernel\n", base, dv, base + dv);
	}
	*/
	new_volt[ id ] = base + dv;
}

void do_CalcVolt(double* new_volt, double* volt, double* current, int* xpos, int* xneg,
				 int* ypos, int* yneg, int* zpos, int* zneg,
				 double DD, double dt, int number)
{
	int blockDim = 128;
	int gridDim = number/blockDim;
	
	gridDim = gridDim * blockDim < number ? gridDim + 1 : gridDim;
	assert(gridDim * blockDim >= number);
	hipDeviceSynchronize();
	
	VoltKernel<<< gridDim, blockDim >>>(new_volt, volt, current, xpos, xneg,
										ypos, yneg, zpos, zneg, DD, dt, number);
	hipDeviceSynchronize();
}
