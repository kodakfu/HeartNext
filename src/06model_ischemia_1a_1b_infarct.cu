#include "hip/hip_runtime.h"
//MyEmail:xsjshr1108@163.com
#include"base.cuh"
#include<stdio.h>
const int ENDO = 1;//normal 1,2,3;infarct 4,5,6;ischemia1a 7,8,9;ischemia1b 10,11,12;
const int MCELL = 2;
const int EPI = 3;
const int ENDO_S = 4;
const int MCELL_S = 5;
const int EPI_S = 6;
const int ENDO_S_1A = 7;
const int MCELL_S_1A = 8;
const int EPI_S_1A = 9;
const int ENDO_S_1B = 10;
const int MCELL_S_1B = 11;
const int EPI_S_1B = 12;
__global__ void kernel06(double* Cai_buffer, double* CaSR_buffer, double* CaSS_buffer, double* Nai_buffer,
						 double* Ki_buffer,  double* sm_buffer, double* sh_buffer, double* sj_buffer,
						 double* sxr1_buffer, double* sxr2_buffer,double* sxs_buffer,double* sr_buffer,
						 double* ss_buffer, double* sd_buffer, double* sf_buffer, double* sf2_buffer,
						 double* sfcass_buffer, double* sOO_buffer, double* sRR_buffer, double* sml_buffer,
						 double* shl_buffer, double* volt_buffer, double* istim_buffer, double* itot_buffer,
						 int* type_buffer, int cellNumber)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index >= cellNumber) return;
	
	double &V = volt_buffer[index];
	if(V == 15.0) V = 15.001;
	double &Istim = istim_buffer[index];
	double &out_itot = itot_buffer[index];
	const int type = type_buffer[index];
	// cell param begin
	double &Cai =Cai_buffer[index] ; 
	double &CaSR=CaSR_buffer[index]; 
	double &CaSS=CaSS_buffer[index]; 
	double &Nai =Nai_buffer[index] ; 
	double &Ki  =Ki_buffer[index]  ; 

	double &sm   =sm_buffer[index]   ; 
	double &sh   =sh_buffer[index]   ; 
	double &sj   =sj_buffer[index]   ; 

	double &sxr1 =sxr1_buffer[index] ; 
	double &sxr2 =sxr2_buffer[index] ; 
							    
	double &sxs  =sxs_buffer[index]  ; 
	double &sr   =sr_buffer[index]   ; 
	double &ss   =ss_buffer[index]   ; 
							    
	double &sd   =sd_buffer[index]   ; 
	double &sf   =sf_buffer[index]   ; 
	double &sf2  =sf2_buffer[index]  ; 
	double &sfcass =sfcass_buffer[index] ;
	double &sOO  =sOO_buffer[index] ;
	double &sRR  =sRR_buffer[index] ;

	double &sml=sml_buffer[index];
	double &shl=shl_buffer[index];
	// cell param end

	double INa_l=0;  ///////////////////////////////////////////////
	double INaL=0;
	double Gna_l = 0.0065;//////////////////////////////////////////
	double alpham_nal =0;///////////////////////////////////////////
	double betam_nal=0;/////////////////////////////////////////////
	double minf_nal = 0;////////////////////////////////////////////
	double hinf_nal=0;//////////////////////////////////////////////
	double taum_nal = 0;////////////////////////////////////////////
	double tauh_nal=600.0;//////////////////////////////////////////
                
	const double Cao=2.0; ////luo
	const double Nao=140.0; ////luo
	const double Vc= 0.016404; ////luo-change
	const double Vsr=0.001094; ////luo
	const double Vss=0.00005468;////luo
		
	const double Bufc=0.2; ////luo
	const double Kbufc=0.001; ////luo
	const double Bufsr=10; ////luo
	const double Kbufsr=0.3; ////luo
	const double Bufss=0.4;////luo
	const double Kbufss=0.00025;////luo
	const double Vmaxup=0.006375;////luo
	const double Kup=0.00025;////luo
	const double Vrel=0.102;////luo
	const double k1_=0.15;////luo
	const double k2_=0.045;////luo
	const double k3=0.060;////luo
	const double k4=0.005;////luo
	const double EC=1.5;////luo
	const double maxsr=2.5;////luo
	const double minsr=1.;////luo
	const double Vleak=0.00036; ////luo
	const double Vxfer=0.0038;////luo
		
	const double R=8314.472;////luo
	const double F=96486.3415;////luo-change
	const double T=310.0;////luo
	const double RTONF=(R*T)/F;////luo
		    
	const double CAPACITANCE=0.185; ////luo
        double Gkr=0.153;
	const double pKNa=0.03; ////luo
	const double GK1=5.405; ////luo
        double GNa=14.838; 
	const double GbNa=0.00029; ////luo
	const double KmK=1.0; ////luo
	const double KmNa=40.0; ////luo
	const double knak=2.724; ////luo
	double GCaL=0.00003980;
	const double GbCa=0.000592;////luo  
	const double knaca=1000; ////luo
	const double KmNai=87.5;////luo
	const double KmCa=1.38; ////luo
	const double ksat=0.1;////luo
	const double n=0.35;////luo
	const double GpCa=0.1238; ////luo
	const double KpCa=0.0005;////luo
	const double GpK=0.0146;////luo

	const double  dt=0.02;

	const double inverseVcF2=1/(2*Vc*F);////luo
	const double inverseVcF=1./(Vc*F);////luo
	const double inversevssF2=1/(2*Vss*F);////luo

	double Ko=5.4;////luo
	if(type== ENDO_S_1A ||type== EPI_S_1A||type==MCELL_S_1A||type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B) Ko=8;//liang_change2
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) Ko=8;//liang_change2
	double Gks, Gto;////luo

	double IKr=0,IKs=0,IK1=0,Ito=0,INa=0,IbNa=0,ICaL=0,IbCa=0,INaCa=0,IpCa=0,IpK=0,INaK=0,Irel=0,Ileak=0,Iup=0,Ixfer=0,IKATP=0;////luo
	double k1=0,k2=0,kCaSR=0,dNai=0,dKi=0,dCai=0,dCaSR=0,dCaSS=0,dRR=0;////luo
	double Ek=0,Ena=0,Eks=0,Eca=0;////luo
	double CaCSQN=0,bjsr=0,cjsr=0,CaSSBuf=0,bcss=0,ccss=0,CaBuf=0,bc=0,cc=0,Ak1=0,Bk1=0,rec_iK1=0,rec_ipK=0,rec_iNaK=0,AM=0,BM=0,////luo
		AH_1=0,BH_1=0,AH_2=0,BH_2=0,AJ_1=0,BJ_1=0,AJ_2=0,BJ_2=0,M_INF=0,H_INF=0,J_INF=0,TAU_M=0,TAU_H=0,TAU_J=0,axr1=0,bxr1=0,axr2=0,bxr2=0,////luo
		Xr1_INF=0,Xr2_INF=0,TAU_Xr1=0,TAU_Xr2=0,Axs=0,Bxs=0,Xs_INF=0,TAU_Xs=0,R_INF=0,TAU_R=0,S_INF=0,TAU_S=0,Ad=0,Bd=0,Cd=0,Af=0,Bf=0,Cf=0,////luo
		Af2=0,Bf2=0,Cf2=0,TAU_D=0,D_INF=0,TAU_F=0,F_INF=0,TAU_F2=0,F2_INF=0,TAU_FCaSS=0,FCaSS_INF=0;////luo
	// calculate start
	Ek=RTONF*(log((Ko/Ki)));////luo
	Ena=RTONF*(log((Nao/Nai)));////luo
	Eks=RTONF*(log((Ko+pKNa*Nao)/(Ki+pKNa*Nai)));////luo
	Eca=0.5*RTONF*(log((Cao/Cai)));////luo

	Ak1=0.1/(1.+exp(0.06*(V-Ek-200)));////luo
	Bk1=(3.*exp(0.0002*(V-Ek+100))+exp(0.1*(V-Ek-10)))/(1.+exp(-0.5*(V-Ek)));////luo

	rec_iK1=Ak1/(Ak1+Bk1);////luo
	rec_iNaK=(1./(1.+0.1245*exp(-0.1*V*F/(R*T))+0.0353*exp(-V*F/(R*T))));////luo
	rec_ipK=1./(1.+exp((25-V)/5.98));////luo

	if(type == EPI||type == ENDO||type == EPI_S||type == ENDO_S)   //////////////////////////////////////////////
		INa_l = Gna_l*sml*sml*sml*shl*(V-Ena);/////////////////////////////////////////////
	if(type == MCELL||type == MCELL_S){////////////////////////////////////////////////////////////////
		INa_l = 1.46*Gna_l*sml*sml*sml*shl*(V-Ena);////////////////////////////////////////
		//GCaL = GCaL * 6.0;
		//Gkr = Gkr * 0.2;
	}
	INa_l =0;

	double mL=0;
	double hL=1;
	double mLss=1.0/(1.0+exp((-(V+42.85))/5.264));
	double tmL=1.0/(6.765*exp((V+11.64)/34.77)+8.552*exp(-(V+77.42)/5.955));
	mL=mLss-(mLss-mL)*exp(-dt/tmL);
	double hLss=1.0/(1.0+exp((V+87.61)/7.488));
	double thL=200.0;
	hL=hLss-(hLss-hL)*exp(-dt/thL);
	double GNaL=0.0075;
	INaL=GNaL*(V-Ena)*mL*hL;
	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) INaL=INaL*1.5;
	
	double ATPi=4.6;              //liang_change
    	double GKATP=3.9;                                               //liang_change
   	double H=2.0;                                                   //liang_change
    	double n1=0.24;                                                  //liang_change
	double Khalf=0.25;  
        IKATP=GKATP*(1/(1+pow(ATPi/Khalf,H)))*pow(Ko/5.4,n1)*(V-Ek);//liang_change
	if(type== ENDO||type== EPI||type==MCELL) IKATP=0;//liang_change2




	INa=GNa*sm*sm*sm*sh*sj*(V-Ena);////luo
        	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) INa=INa*0.887;
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) INa=INa*0.38;
         
	ICaL=GCaL*sd*sf*sf2*sfcass*4*(V-15)*(F*F/(R*T))*(0.25*exp(2*(V-15)*F/(R*T))*CaSS-Cao)/(exp(2*(V-15)*F/(R*T))-1.);////luo
	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) ICaL=ICaL*0.8;
	if(type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B) ICaL=ICaL*0.5;
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) ICaL=ICaL*0.62;

	if(type== ENDO||type== ENDO_S||type== ENDO_S_1A||type== ENDO_S_1B) Gto=0.073;////luo
	if(type== MCELL||type== EPI||type== MCELL_S||type== EPI_S||type== MCELL_S_1A||type== EPI_S_1A||type== MCELL_S_1B||type== EPI_S_1B)  Gto=0.294;////luo
	Ito=Gto*sr*ss*(V-Ek);////luo
    	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) Ito=Ito*0.5;
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) Ito=Ito*0.37;

	IKr=Gkr*sqrt(Ko/5.4)*sxr1*sxr2*(V-Ek);////luo
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) IKr=IKr*0.3;
	
        if(type== ENDO ||type== EPI||type== ENDO_S ||type== EPI_S||type== ENDO_S_1A ||type== EPI_S_1A||type== ENDO_S_1B ||type== EPI_S_1B) Gks=0.392;   //liang_change  //liang_change 
	if(type== MCELL||type== MCELL_S||type== MCELL_S_1A||type== MCELL_S_1B) Gks=0.098;   //liang_change  //liang_change 
	IKs=Gks*sxs*sxs*(V-Eks);////luo
	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) IKs=IKs*0.781;
	if(type== ENDO_S||type== EPI_S||type==MCELL_S) IKs=IKs*0.2;

	IK1=GK1*sqrt(Ko/5.4)*rec_iK1*(V-Ek);////luo
		
	INaCa=knaca*(1./(KmNai*KmNai*KmNai+Nao*Nao*Nao))*(1./(KmCa+Cao))*(1./(1+ksat*exp((n-1)*V*F/(R*T))))*(exp(n*V*F/(R*T))*Nai*Nai*Nai*Cao-exp((n-1)*V*F/(R*T))*Nao*Nao*Nao*Cai*2.5);////luo
	if(type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B) INaCa=INaCa*0.6;

	INaK=knak*(Ko/(Ko+KmK))*(Nai/(Nai+KmNa))*rec_iNaK;////luo
	if(type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B) INaK=INaK*0.46;

	IpCa=GpCa*Cai/(KpCa+Cai);////luo
	IpK=GpK*rec_ipK*(V-Ek);////luo
	IbNa=GbNa*(V-Ena);////luo
	IbCa=GbCa*(V-Eca);////luo
	if(type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B) IbCa=IbCa*1.3;

	
	out_itot=IKr+IKs+IK1+Ito+INa+IbNa+ICaL+IbCa+INaK+INaCa+IpCa+IpK+Istim+INaL+IKATP+INa_l;	////luo//liang_change2
	
	kCaSR=maxsr-((maxsr-minsr)/(1+(EC/CaSR)*(EC/CaSR)));////luo
	k1=k1_/kCaSR;////luo
	k2=k2_*kCaSR;////luo
	dRR=k4*(1-sRR)-k2*CaSS*sRR;////luo
	sRR+=dt*dRR;////luo
	sOO=k1*CaSS*CaSS*sRR/(k3+k1*CaSS*CaSS);////luo

	Irel=Vrel*sOO*(CaSR-CaSS);////luo
	Ileak=Vleak*(CaSR-Cai);////luo
	Iup=Vmaxup/(1.+((Kup*Kup)/(Cai*Cai)));////luo
	Ixfer=Vxfer*(CaSS-Cai);////luo
	if(type== ENDO_S_1B||type== EPI_S_1B||type==MCELL_S_1B)
	{
	Irel=Irel*0.65;
	Iup=Iup*0.71;	
	}

	CaCSQN=Bufsr*CaSR/(CaSR+Kbufsr);////luo
	dCaSR=dt*(Iup-Irel-Ileak);////luo
	bjsr=Bufsr-CaCSQN-dCaSR-CaSR+Kbufsr;////luo
	cjsr=Kbufsr*(CaCSQN+dCaSR+CaSR);////luo
	CaSR=(sqrt(bjsr*bjsr+4*cjsr)-bjsr)/2; ////luo

	CaSSBuf=Bufss*CaSS/(CaSS+Kbufss);////luo
	dCaSS=dt*(-Ixfer*(Vc/Vss)+Irel*(Vsr/Vss)+(-ICaL*inversevssF2*CAPACITANCE));////luo
	bcss=Bufss-CaSSBuf-dCaSS-CaSS+Kbufss;////luo
	ccss=Kbufss*(CaSSBuf+dCaSS+CaSS);////luo
	CaSS=(sqrt(bcss*bcss+4*ccss)-bcss)/2;////luo

	CaBuf=Bufc*Cai/(Cai+Kbufc);////luo
	dCai=dt*((-(IbCa+IpCa-2*INaCa)*inverseVcF2*CAPACITANCE)-(Iup-Ileak)*(Vsr/Vc)+Ixfer);////luo
	bc=Bufc-CaBuf-dCai-Cai+Kbufc;////luo
	cc=Kbufc*(CaBuf+dCai+Cai);////luo
	Cai=(sqrt(bc*bc+4*cc)-bc)/2;////luo
    
	dNai=-(INa+INaL+IbNa+3*INaK+3*INaCa)*inverseVcF*CAPACITANCE;////luo
	Nai+=dt*dNai;////luo
    
	dKi=-(Istim+IK1+Ito+IKr+IKs+IKATP-2*INaK+IpK)*inverseVcF*CAPACITANCE;////luo
	Ki+=dt*dKi;////luo
 
	AM=1./(1.+exp((-60.-V)/5.));////luo
	BM=0.1/(1.+exp((V+35.)/5.))+0.10/(1.+exp((V-50.)/200.));////luo
	TAU_M=AM*BM;////luo
	M_INF=1./((1.+exp((-56.86-V)/9.03))*(1.+exp((-56.86-V)/9.03)));////luo
	if(type== ENDO_S_1A||type== EPI_S_1A||type==MCELL_S_1A) 
	M_INF=1./((1.+exp((-55.5-V)/9.03))*(1.+exp((-55.5-V)/9.03)));

	if(V>=-40.)
		{
			AH_1=0.;////luo
			BH_1=(0.77/(0.13*(1.+exp(-(V+10.66)/11.1))));////luo
			TAU_H=1.0/(AH_1+BH_1);////luo
		}
	else
		{
			AH_2=(0.057*exp(-(V+80.)/6.8));////luo
			BH_2=(2.7*exp(0.079*V)+(3.1e5)*exp(0.3485*V));////luo
			TAU_H=1.0/(AH_2+BH_2);////luo
		}
	H_INF=1./((1.+exp((V+71.55)/7.43))*(1.+exp((V+71.55)/7.43)));////luo
	if(V>=-40.)
		{
			AJ_1=0.;////luo
			BJ_1=(0.6*exp((0.057)*V)/(1.+exp(-0.1*(V+32.))));////luo
			TAU_J= 1.0/(AJ_1+BJ_1);////luo
		}
	else
		{
			AJ_2=(((-2.5428e4)*exp(0.2444*V)-(6.948e-6)*exp(-0.04391*V))*(V+37.78)/(1.+exp(0.311*(V+79.23))));////luo
			BJ_2=(0.02424*exp(-0.01052*V)/(1.+exp(-0.1378*(V+40.14))));////luo
			TAU_J= 1.0/(AJ_2+BJ_2);////luo
		}
	J_INF=H_INF;////luo


	alpham_nal = 0.32*(V+47.13)/(1.0 - exp(-0.1*(V+47.13)));//////////////////////////////////
	betam_nal = 0.08*exp(V/11.0);/////////////////////////////////////////////////////////////
	minf_nal = alpham_nal/(alpham_nal+betam_nal);/////////////////////////////////////////////
	taum_nal = 1.0/(alpham_nal+betam_nal);////////////////////////////////////////////////////
	hinf_nal = 1.0/(1.0 + exp((V+91.0)/6.1));/////////////////////////////////////////////////
	tauh_nal= 600.0;//////////////////////////////////////////////////////////////////////////

	Xr1_INF=1./(1.+exp((-26.-V)/7.));////luo
	axr1=450./(1.+exp((-45.-V)/10.));////luo
	bxr1=6./(1.+exp((V-(-30.))/11.5));////luo
	TAU_Xr1=axr1*bxr1;////luo
	Xr2_INF=1./(1.+exp((V-(-88.))/24.));////luo
	axr2=3./(1.+exp((-60.-V)/20.));////luo
	bxr2=1.12/(1.+exp((V-60.)/20.));////luo
	TAU_Xr2=axr2*bxr2;////luo

	Xs_INF=1./(1.+exp((-5.-V)/14.));////luo
	//Axs = 1400. * rsqrt(1. + exp((5.-V)/6) );
	Axs=(1400./(sqrt(1.+exp((5.-V)/6))));////luo
	Bxs=(1./(1.+exp((V-35.)/15.)));////luo
	TAU_Xs=Axs*Bxs+80;////luo
	
	if(type== EPI||type== EPI_S||type== EPI_S_1B||type== MCELL||type== MCELL_S||type== MCELL_S_1B)
		{
			R_INF=1./(1.+exp((20-V)/6.));////luo
			S_INF=1./(1.+exp((V+20)/5.));////luo
			TAU_R=9.5*exp(-(V+40.)*(V+40.)/1800.)+0.8;////luo
			TAU_S=85.*exp(-(V+45.)*(V+45.)/320.)+5./(1.+exp((V-20.)/5.))+3.;////luo
		}
	if(type== ENDO||type== ENDO_S||type== ENDO_S_1B)
		{
			R_INF=1./(1.+exp((20-V)/6.));////luo
			S_INF=1./(1.+exp((V+28)/5.));////luo
			TAU_R=9.5*exp(-(V+40.)*(V+40.)/1800.)+0.8;////luo
			TAU_S=1000.*exp(-(V+67)*(V+67)/1000.)+8.;////luo
		}
	if(type== EPI_S_1A||type== MCELL_S_1A)
		{
			R_INF=1./(1.+exp((27.2-V)/6.));////luo
			S_INF=1./(1.+exp((V+6.3)/5.));////luo
			TAU_R=9.5*exp(-(V+40.)*(V+40.)/1800.)+0.8;////luo
			TAU_S=85.*exp(-(V+45.)*(V+45.)/320.)+5./(1.+exp((V-20.)/5.))+3.;////luo
		}
	if(type== ENDO_S_1A)
		{
			R_INF=1./(1.+exp((27.2-V)/6.));////luo
			S_INF=1./(1.+exp((V+14.3)/5.));////luo
			TAU_R=9.5*exp(-(V+40.)*(V+40.)/1800.)+0.8;////luo
			TAU_S=1000.*exp(-(V+67)*(V+67)/1000.)+8.;////luo
		}

	D_INF=1./(1.+exp((-8-V)/7.5));////luo
	Ad=1.4/(1.+exp((-35-V)/13))+0.25;////luo
	Bd=1.4/(1.+exp((V+5)/5));////luo
	Cd=1./(1.+exp((50-V)/20));////luo
	TAU_D=Ad*Bd+Cd;////luo
	F_INF=1./(1.+exp((V+20)/7));////luo
	Af=1102.5*exp(-(V+27)*(V+27)/225);////luo
	Bf=200./(1+exp((13-V)/10.));////luo
	Cf=(180./(1+exp((V+30)/10)))+20;////luo
	TAU_F=(Af+Bf+Cf);////luo                    
	F2_INF=0.67/(1.+exp((V+35)/7))+0.33;////luo
	Af2=600*exp(-(V+25)*(V+25)/170);////luo
	Bf2=31/(1.+exp((25-V)/10));////luo
	Cf2=16/(1.+exp((V+30)/10));////luo
	TAU_F2=Af2+Bf2+Cf2;////luo
	FCaSS_INF=0.6/(1+(CaSS/0.05)*(CaSS/0.05))+0.4;////luo
	TAU_FCaSS=80./(1+(CaSS/0.05)*(CaSS/0.05))+2.;////luo



	sml = minf_nal - (minf_nal-sml)*exp(-dt/taum_nal);////////////////////////////////////////////////////
	shl = hinf_nal - (hinf_nal-shl)*exp(-dt/tauh_nal);////////////////////////////////////////////////////
		
	sm=M_INF-(M_INF-sm)*exp(-dt/TAU_M);////luo
	sh=H_INF-(H_INF-sh)*exp(-dt/TAU_H);////luo
	sj=J_INF-(J_INF-sj)*exp(-dt/TAU_J);////luo
	sxr1=Xr1_INF-(Xr1_INF-sxr1)*exp(-dt/TAU_Xr1);////luo
	sxr2=Xr2_INF-(Xr2_INF-sxr2)*exp(-dt/TAU_Xr2);////luo
	sxs=Xs_INF-(Xs_INF-sxs)*exp(-dt/TAU_Xs);////luo
	ss=S_INF-(S_INF-ss)*exp(-dt/TAU_S);////luo
	sr=R_INF-(R_INF-sr)*exp(-dt/TAU_R);////luo
	sd =D_INF-(D_INF-sd)*exp(-dt/TAU_D);////luo
	sf=F_INF-(F_INF-sf)*exp(-dt/TAU_F);////luo
	sf2=F2_INF-(F2_INF-sf2)*exp(-dt/TAU_F2);////luo
	sfcass=FCaSS_INF-(FCaSS_INF-sfcass)*exp(-dt/TAU_FCaSS);////luo
}

void call_kernel06(double* Cai_buffer, double* CaSR_buffer, double* CaSS_buffer, double* Nai_buffer,
				   double* Ki_buffer,  double* sm_buffer, double* sh_buffer, double* sj_buffer,
				   double* sxr1_buffer, double* sxr2_buffer,double* sxs_buffer,double* sr_buffer,
				   double* ss_buffer, double* sd_buffer, double* sf_buffer, double* sf2_buffer,
				   double* sfcass_buffer, double* sOO_buffer, double* sRR_buffer, double* sml_buffer,
				   double* shl_buffer, double* volt_buffer, double* istim_buffer, double* itot_buffer,
				   int* type_buffer, int cellNumber, int threadCount)
{
	int blockDim = threadCount;
	int gridDim = cellNumber/blockDim;
	gridDim = gridDim * blockDim < cellNumber ? gridDim + 1 : gridDim;
	/*
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	*/
	kernel06<<< gridDim, blockDim >>>
		( Cai_buffer,  CaSR_buffer,  CaSS_buffer,  Nai_buffer,
		  Ki_buffer,  sm_buffer,  sh_buffer,  sj_buffer,
		  sxr1_buffer,  sxr2_buffer,  sxs_buffer,  sr_buffer,
		  ss_buffer,  sd_buffer,  sf_buffer,  sf2_buffer,
		  sfcass_buffer,  sOO_buffer,  sRR_buffer,  sml_buffer,
		  shl_buffer,  volt_buffer,  istim_buffer,  itot_buffer,
		  type_buffer, cellNumber);
	//hipDeviceSynchronize();
	/*
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	printf("%fms\n", time);
	hipEventDestroy( start );
	hipEventDestroy( stop );
	*/
}

